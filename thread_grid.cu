
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    printf("Thread x:%d thread y:%d thread z:%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("Block x:%d block y:%d block z:%d\n", blockIdx.x, blockIdx.y, blockIdx.z);
}

int main()
{
    unsigned int nx = 16;
    unsigned int ny = 16;
    
    dim3 block(8, 8, 8);
    dim3 grid( nx / block.x, ny / block.y);
    
    helloCUDA<<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}
